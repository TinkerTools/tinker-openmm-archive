#include "hip/hip_runtime.h"
float2 bondParams = PARAMS[index];
real deltaIdeal = r-bondParams.x;
energy += 0.5f * bondParams.y*deltaIdeal*deltaIdeal;
real dEdR = bondParams.y * deltaIdeal;
real dE = dEdR/r;
real3 ab = make_real3(POSQ[atom1].x - POSQ[atom2].x,POSQ[atom1].y- POSQ[atom2].y, POSQ[atom1].z - POSQ[atom2].z);
#if APPLY_PERIODIC
APPLY_PERIODIC_TO_DELTA(ab);  
#endif
real dEdx = dE * ab.x;
real dEdy = dE * ab.y;
real dEdz = dE * ab.z;
vxx+= ab.x * dEdx;
vxy+= ab.y * dEdx;
vxz+= ab.z * dEdx;
vyy+= ab.y * dEdy;
vyz+= ab.z * dEdy;
vzz+= ab.z * dEdz;
